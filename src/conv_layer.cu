#include "hip/hip_runtime.h"
#include "conv_layer.hpp"
#include <cmath>
#include <random>
#include <ctime>

ConvLayer::ConvLayer(int batch_size_, int in_channels_, int in_hight_, 
    int in_width_, int out_channels_, int kernel_size_, int stride_, 
    int padding_) {
    batch_size = batch_size_;
    in_channels = in_channels_;
    in_height = in_hight_;
    in_width = in_width_;
    out_channels = out_channels_;
    kernel_size = kernel_size_;
    stride = stride_;
    padding = padding_;
}

ConvLayer::~ConvLayer() {
    // Destroy tensor descriptors
    checkCUDNN(hipdnnDestroyTensorDescriptor(src_tensor_desc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(dst_tensor_desc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(bias_tensor_desc));
    checkCUDNN(hipdnnDestroyFilterDescriptor(filter_desc));
    // Destroy conv layer descriptors
    checkCUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
    // Free GPU Memory
    checkCudaErrors(hipFree(weight));
    checkCudaErrors(hipFree(bias));
    checkCudaErrors(hipFree(d_weight));
    checkCudaErrors(hipFree(d_bias));
    checkCudaErrors(hipFree(bottom_diff));
    checkCudaErrors(hipFree(data_out));
}

void ConvLayer::SetDescriptor() {
    // Set input, bias and filter descriptor
    checkCUDNN(hipdnnSetTensor4dDescriptor(src_tensor_desc, HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_FLOAT, batch_size, in_channels, in_height, in_width));
    checkCUDNN(hipdnnSetTensor4dDescriptor(bias_tensor_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, 1, out_channels, 1, 1));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT,
        HIPDNN_TENSOR_NCHW, out_channels, in_channels, kernel_size, kernel_size));
    // Set conv layer descriptor
    const int padA[2] = {padding, padding};
    const int filterStrideA[2] = {stride, stride};
    const int upscaleA[2] = {1, 1};
    checkCUDNN(hipdnnSetConvolutionNdDescriptor(conv_desc, 2, padA, filterStrideA,
        upscaleA, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
    // Set output tensor descriptor
    // Use cudnnGetConvolutionNdForwardOutputDim to avoid computing by myself
    int tensorOuputDimA[4];
    checkCUDNN(cudnnGetConvolutionNdForwardOutputDim(conv_desc, src_tensor_desc, 
        filter_desc, 4, tensorOuputDimA));
    checkCUDNN(hipdnnSetTensor4dDescriptor(dst_tensor_desc, HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_FLOAT, tensorOuputDimA[0], tensorOuputDimA[1], 
        tensorOuputDimA[2], tensorOuputDimA[3]));
    out_size = tensorOuputDimA[0] * tensorOuputDimA[1] * tensorOuputDimA[2] * 
        tensorOuputDimA[3];
}

void ConvLayer::SetAlgorithm(hipdnnHandle_t *handle) {
    // Set forward algorithm
    int requestedAlgoCount = HIPDNN_CONVOLUTION_FWD_ALGO_COUNT;
    int returnedAlgoCount = -1;
    hipdnnConvolutionFwdAlgoPerf_t fwd_results[2 * requestedAlgoCount];
    checkCUDNN(hipdnnFindConvolutionForwardAlgorithm(*handle, src_tensor_desc, 
        filter_desc, conv_desc, dst_tensor_desc, requestedAlgoCount, 
        &returnedAlgoCount, fwd_results));
    fwd_algo = fwd_results[0].algo;
    // Set backward algorithm
    // for filter
    requestedAlgoCount = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_COUNT;
    returnedAlgoCount = -1;
    hipdnnConvolutionBwdFilterAlgoPerf_t bwd_filter_results[2 * requestedAlgoCount];
    checkCUDNN(hipdnnFindConvolutionBackwardFilterAlgorithm(*handle, src_tensor_desc, 
        dst_tensor_desc, conv_desc, filter_desc, requestedAlgoCount, 
        &returnedAlgoCount, bwd_filter_results));
    bwd_filter_algo = bwd_filter_results[0].algo;
    // for data
    requestedAlgoCount = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_TRANSPOSE_GEMM;
    returnedAlgoCount = -1;
    hipdnnConvolutionBwdDataAlgoPerf_t bwd_data_results[2 * requestedAlgoCount];
    checkCUDNN(hipdnnFindConvolutionBackwardDataAlgorithm(*handle, filter_desc, 
        dst_tensor_desc, conv_desc, src_tensor_desc, requestedAlgoCount, 
        &returnedAlgoCount, bwd_data_results));
    bwd_data_algo = bwd_data_results[0].algo;
}

void ConvLayer::EvalSet() {
    batch_size = 1;
    SetDescriptor();
    checkCudaErrors(hipFree(data_out));
    checkCudaErrors(hipMalloc((void**)&data_out, sizeof(float) * out_size));
}

void ConvLayer::InitParameter(hipdnnHandle_t *handle) {
    /**
    * Initialization for cudnn settings
    */
    // Create tensor descriptors
    checkCUDNN(hipdnnCreateTensorDescriptor(&src_tensor_desc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&dst_tensor_desc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&bias_tensor_desc));
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_desc));
    // Create conv layer descriptors
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
    // Set descriptors
    SetDescriptor();
    // Set algorithm
    SetAlgorithm(handle);
    /**
    * Initialization for data
    */
    int in_size = batch_size * in_channels * in_height * in_width;
    int weight_size = out_channels * in_channels * kernel_size * kernel_size;
    // Random init weight and bias
    float *host_weight = (float *)malloc(sizeof(float) * weight_size);
    float *host_bias = (float *)malloc(sizeof(float) * out_channels);
    // Init host tensor
    std::default_random_engine e(time(0));
    float wconv = sqrt(3.0f / (kernel_size * kernel_size * in_channels));
    std::uniform_real_distribution<> dconv(-wconv, wconv);
    for (int i = 0; i < weight_size; i++) {
        host_weight[i] = dconv(e);
    }
    for (int i = 0; i < out_channels; i++) {
        host_bias[i] = dconv(e);
    }
    // Copy to device tensor
    checkCudaErrors(hipMalloc((void**)&weight, sizeof(float) * weight_size));
    checkCudaErrors(hipMemcpy(weight, host_weight, sizeof(float) * weight_size, 
        hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&bias, sizeof(float) * out_channels));
    checkCudaErrors(hipMemcpy(bias, host_bias, sizeof(float) * out_channels, 
        hipMemcpyHostToDevice));
    free(host_weight);
    free(host_bias);
    // Allocate memory for data_out and backward diff
    checkCudaErrors(hipMalloc((void**)&d_weight, sizeof(float) * weight_size));
    checkCudaErrors(hipMalloc((void**)&d_bias, sizeof(float) * out_channels));
    checkCudaErrors(hipMalloc((void**)&bottom_diff, sizeof(float) * in_size));
    checkCudaErrors(hipMalloc((void**)&data_out, sizeof(float) * out_size));
}

float* ConvLayer::forward(float *input, hipdnnHandle_t *handle) {
    data_in = input;
    // allocate workspace
    size_t sizeInBytes=0;
    void* workSpace=NULL;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(*handle, src_tensor_desc, 
        filter_desc, conv_desc, dst_tensor_desc, fwd_algo, &sizeInBytes));
    if (sizeInBytes!=0) {
        checkCudaErrors(hipMalloc(&workSpace, sizeInBytes));
    }
    // conv
    float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnConvolutionForward(*handle, &alpha, src_tensor_desc,
        data_in, filter_desc, weight, conv_desc, fwd_algo, workSpace, sizeInBytes, 
        &beta, dst_tensor_desc, data_out));
    // addBias
    checkCUDNN(hipdnnAddTensor(*handle, &alpha, bias_tensor_desc, bias, &alpha,
        dst_tensor_desc, data_out));
    if (sizeInBytes!=0) {
        checkCudaErrors(hipFree(workSpace));
    }
    return data_out;
}

float* ConvLayer::backward(float *top_diff, hipdnnHandle_t *handle) {
    // allocate workspace for cudnnConvolutionBackward
    size_t sizeInBytes = 0;
    size_t sizeTempBytes = 0;
    void* workSpace=NULL;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(*handle, 
        src_tensor_desc, dst_tensor_desc, conv_desc, filter_desc, 
        bwd_filter_algo, &sizeTempBytes));
    sizeInBytes = sizeTempBytes;
    checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(*handle, 
        filter_desc, dst_tensor_desc, conv_desc, src_tensor_desc, 
        bwd_data_algo, &sizeTempBytes));
    sizeInBytes = sizeInBytes > sizeTempBytes ? sizeInBytes : sizeTempBytes;
    if (sizeInBytes!=0) {
        checkCudaErrors(hipMalloc(&workSpace, sizeInBytes));
    }

    float alpha = 1.0f, beta = 0.0f;
    // bias backward
    checkCUDNN(hipdnnConvolutionBackwardBias(*handle, &alpha, dst_tensor_desc, 
        top_diff, &beta, bias_tensor_desc, d_bias));
    // weight backward
    checkCUDNN(hipdnnConvolutionBackwardFilter(*handle, &alpha, src_tensor_desc, 
        data_in, dst_tensor_desc, top_diff, conv_desc, bwd_filter_algo, 
        workSpace, sizeInBytes, &beta, filter_desc, d_weight));
    // data backward
    checkCUDNN(hipdnnConvolutionBackwardData(*handle, &alpha, filter_desc, weight, 
        dst_tensor_desc, top_diff, conv_desc, bwd_data_algo, workSpace, sizeInBytes,
        &beta, src_tensor_desc, bottom_diff));
    if (sizeInBytes!=0) {
        checkCudaErrors(hipFree(workSpace));
    }
    data_in = nullptr;
    return bottom_diff;
}

void ConvLayer::UpdateWeights(float lr, hipblasHandle_t *handle) {
    int weight_size = out_channels * in_channels * kernel_size * kernel_size;
    float alpha = -lr;
    checkCublasErrors(hipblasSaxpy(*handle, weight_size, &alpha, d_weight, 1, 
        weight, 1));
    checkCublasErrors(hipblasSaxpy(*handle, out_channels, &alpha, d_bias, 1, 
        bias, 1));
}