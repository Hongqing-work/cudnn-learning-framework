#include "relu_layer.hpp"

ReluLayer::ReluLayer(int batch_size_, int in_size_) {
    batch_size = batch_size_;
    in_size = in_size_;
}

ReluLayer::~ReluLayer() {
    // Destroy tensor descriptors
    checkCUDNN(hipdnnDestroyTensorDescriptor(src_tensor_desc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(dst_tensor_desc));
    // Destroy activation layer descriptor
    checkCUDNN(hipdnnDestroyActivationDescriptor(activation_desc));
    // Free GPU Memory
    checkCudaErrors(hipFree(bottom_diff));
    checkCudaErrors(hipFree(data_out));
}

void ReluLayer::SetDescriptor() {
    // Set tensor descriptor
    checkCUDNN(hipdnnSetTensor4dDescriptor(src_tensor_desc, HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_FLOAT, batch_size, in_size, 1, 1));
    checkCUDNN(hipdnnSetTensor4dDescriptor(dst_tensor_desc, HIPDNN_TENSOR_NCHW,
        HIPDNN_DATA_FLOAT, batch_size, in_size, 1, 1));
    // Set activation layer descriptor
    checkCUDNN(hipdnnSetActivationDescriptor(activation_desc, 
        HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
}

void ReluLayer::EvalSet() {
    batch_size = 1;
    SetDescriptor();
    checkCudaErrors(hipFree(data_out));
    checkCudaErrors(hipMalloc((void**)&data_out, sizeof(float) * in_size));
}

void ReluLayer::InitParameter() {
    /**
    * Initialization for cudnn settings
    */
    // Create tensor descriptors
    checkCUDNN(hipdnnCreateTensorDescriptor(&src_tensor_desc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&dst_tensor_desc));
    // Create activation layer descriptor
    checkCUDNN(hipdnnCreateActivationDescriptor(&activation_desc));
    // Set descriptors
    SetDescriptor();
    
    /**
    * Initialization for data
    */
    // Allocate memory for data_out and backward diff
    int size = batch_size * in_size;
    checkCudaErrors(hipMalloc((void**)&bottom_diff, sizeof(float) * size));
    checkCudaErrors(hipMalloc((void**)&data_out, sizeof(float) * size));
}

float* ReluLayer::forward(float *input, hipdnnHandle_t *handle) {
    data_in = input;
    float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnActivationForward(*handle, activation_desc, &alpha, 
        src_tensor_desc, data_in, &beta, dst_tensor_desc, data_out));
    return data_out;
}

float* ReluLayer::backward(float *top_diff, hipdnnHandle_t *handle) {
    float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnActivationBackward(*handle, activation_desc, &alpha, 
        dst_tensor_desc, data_out, dst_tensor_desc, top_diff, src_tensor_desc, 
        data_in, &beta, src_tensor_desc, bottom_diff));
    return bottom_diff;
}