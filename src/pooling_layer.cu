#include "pooling_layer.hpp"

PoolingLayer::PoolingLayer(int batch_size_, int in_channels_, int in_hight_, 
    int in_width_, int kernel_size_, int stride_, int padding_) {
    batch_size = batch_size_;
    in_channels = in_channels_;
    in_height = in_hight_;
    in_width = in_width_;
    kernel_size = kernel_size_;
    stride = stride_;
    padding = padding_;
}

PoolingLayer::~PoolingLayer() {
    // Destroy tensor descriptors
    checkCUDNN(hipdnnDestroyTensorDescriptor(src_tensor_desc));
    checkCUDNN(hipdnnDestroyTensorDescriptor(dst_tensor_desc));
    // Destroy pooling layer descriptor
    checkCUDNN(hipdnnDestroyPoolingDescriptor(pooling_desc));
    // Free GPU Memory
    checkCudaErrors(hipFree(bottom_diff));
    checkCudaErrors(hipFree(data_out));
}

void PoolingLayer::SetDescriptor() {
    // Set input tensor descriptor
    checkCUDNN(hipdnnSetTensor4dDescriptor(src_tensor_desc, HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_FLOAT, batch_size, in_channels, in_height, in_width));
    // Set pooling layer descriptor
    const int windowDimA[2] = {kernel_size, kernel_size};
    const int paddingA[2] = {padding, padding};
    const int strideA[2] = {stride, stride};
    checkCUDNN(hipdnnSetPoolingNdDescriptor(pooling_desc, HIPDNN_POOLING_MAX,
        HIPDNN_PROPAGATE_NAN, 2, windowDimA, paddingA, strideA));
    // Set output tensor descriptor
    // Use cudnnGetPoolingNdForwardOutputDim to avoid computing by myself
    int tensorOuputDimA[4];
    checkCUDNN(cudnnGetPoolingNdForwardOutputDim(pooling_desc, src_tensor_desc, 
        4, tensorOuputDimA));
    checkCUDNN(hipdnnSetTensor4dDescriptor(dst_tensor_desc, HIPDNN_TENSOR_NCHW, 
        HIPDNN_DATA_FLOAT, tensorOuputDimA[0], tensorOuputDimA[1], 
        tensorOuputDimA[2], tensorOuputDimA[3]));
    out_size = tensorOuputDimA[0] * tensorOuputDimA[1] * tensorOuputDimA[2] * 
        tensorOuputDimA[3];
}

void PoolingLayer::EvalSet() {
    batch_size = 1;
    SetDescriptor();
    checkCudaErrors(hipFree(data_out));
    checkCudaErrors(hipMalloc((void**)&data_out, sizeof(float) * out_size));
}

void PoolingLayer::InitParameter() {
    /**
    * Initialization for cudnn settings
    */
    // Create tensor descriptors
    checkCUDNN(hipdnnCreateTensorDescriptor(&src_tensor_desc));
    checkCUDNN(hipdnnCreateTensorDescriptor(&dst_tensor_desc));
    // Create pooling layer descriptor
    checkCUDNN(hipdnnCreatePoolingDescriptor(&pooling_desc));
    // Set descriptors
    SetDescriptor();

    /**
    * Initialization for data
    */
    int in_size = batch_size * in_channels * in_height * in_width;
    // Allocate memory for data_out and backward diff
    checkCudaErrors(hipMalloc((void**)&bottom_diff, sizeof(float) * in_size));
    checkCudaErrors(hipMalloc((void**)&data_out, sizeof(float) * out_size));
}

float* PoolingLayer::forward(float *input, hipdnnHandle_t *handle) {
    data_in = input;
    float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnPoolingForward(*handle, pooling_desc, &alpha, src_tensor_desc, 
        data_in, &beta, dst_tensor_desc, data_out));
    return data_out;
}

float* PoolingLayer::backward(float *top_diff, hipdnnHandle_t *handle) {
    float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnPoolingBackward(*handle, pooling_desc, &alpha, dst_tensor_desc, 
        data_out, dst_tensor_desc, top_diff, src_tensor_desc, data_in, &beta, 
        src_tensor_desc, bottom_diff));
    return bottom_diff;
}